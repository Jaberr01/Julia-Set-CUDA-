#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include "EasyBMP.h"
#include "EasyBMP.cpp"

//Complex number definition
 struct Complex {	// typedef is not required for C++
	float x; 		// real part is represented on x-axis in output image
	float y; 		// imaginary part is represented by y-axis in output image
};

//Function declarations
void compute_julia(const char*, int, int);
void save_image(uchar4*, const char*, int, int);
__device__ Complex add(Complex, Complex);
__device__ Complex mul(Complex, Complex);
__device__ float mag(Complex);
__global__ void Kernel(uchar4* , int , int , float , float , float , float , float , float , int , int , Complex );

//main function
int main(void) {
	char* name = "test.bmp";
	compute_julia(name, 3000, 3000);	//width x height
	printf("Finished creating %s.\n", name);
	return 0;
}
__global__ void Kernel(uchar4* pixels, int width, int height, float w, float h, float x_min, float x_incr, float y_min, float y_incr, int max_iterations, int infinity, Complex c) {
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < width && col < height) {
	
		Complex z;
		z.x = x_min + col * x_incr;
		z.y = y_min + row * y_incr;
		//iteratively compute z = z^2 + c and check if z goes to infinity
		int n = 0;
		do {
			z = add(mul(z, z), c);								// z = z^2 + c
		} while (mag(z) < infinity && n++ < max_iterations);	// keep looping until z->infinity or we reach max_iterations

		// color each pixel based on above loop
		if (n == max_iterations) {								// if we reach max_iterations before z reaches infinity, pixel is black 
			pixels[col + row * width] = { 0,0,0,0 };
		}
		else {	
			unsigned char h1, h2, h3;
			double shift = 0.4;
			if (n <= shift*max_iterations / 3) {
				h1 = (unsigned char)(255 * sqrt((float)n / max_iterations));
				h2 = 25;
				h3 = 25;

			}
			else if (n <= 2 * shift * max_iterations / 3) {
				h1 = 25;
				h2 = (unsigned char)(255 * sqrt((float)n / max_iterations));
				h3 = 25;
			}
			else {
				h1 = 25;
				h2 = 25;
				h3 = (unsigned char)(255 * sqrt((float)n / max_iterations));
			}
			pixels[col + row * width] = { h1,h2,h3,255 };
		}
	}
		
	
}

// serial implementation of Julia set
void compute_julia(const char* filename, int width, int height) {
	//create output image
	unsigned int N = width * height;

	//PROBLEM SETTINGS (marked by '******')
	// **** Accuracy ****: lower values give less accuracy but faster performance
	int max_iterations = 400;
	int infinity = 20;													//used to check if z goes towards infinity

	// ***** Shape ****: other values produce different patterns. See https://en.wikipedia.org/wiki/Julia_set
	Complex c = { -0.8, 0.156 }; 										//the constant in z = z^2 + c

	// ***** Size ****: higher w means smaller size
	float w = 4;
	float h = w * height / width;										//preserve aspect ratio

	// LIMITS for each pixel
	float x_min = -w / 2, y_min = -h / 2;
	float x_incr = w / width, y_incr = h / height;

	uchar4* pixels;	//uchar4 is a CUDA type representing a vector of 4 chars
	hipMallocManaged(&pixels, N * sizeof(uchar4));

	dim3 blockdim(32,32); //number of threads per block
	dim3 griddim((width + blockdim.x - 1) / blockdim.x, (height + blockdim.y - 1) / blockdim.y); //number of threads per block

	Kernel <<<griddim, blockdim >>> (pixels, width, height, w, h, x_min, x_incr, y_min, y_incr, max_iterations, infinity,c);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		printf("Error: %s\n", hipGetErrorString(err));
	}
	hipDeviceSynchronize();


	//Write output image to a file (DO NOT parallelize this function)
	save_image(pixels, filename, width, height);

	//free memory
	hipFree(pixels);
}

void save_image(uchar4* pixels, const char* filename, int width, int height) {
	BMP output;
	output.SetSize(width, height);
	output.SetBitDepth(24);
	// save each pixel to output image
	for (int row = 0; row < height; row++) {
		for (int col = 0; col < width; col++) {
			uchar4 color = pixels[col + row * width];
			output(col, row)->Red = color.x;
			output(col, row)->Green = color.y;
			output(col, row)->Blue = color.z;
		}
	}
	output.WriteToFile(filename);
}

__device__ Complex add(Complex c1, Complex c2) {
	return{ c1.x + c2.x, c1.y + c2.y };
}

__device__ Complex mul(Complex c1, Complex c2) {
	return{ c1.x * c2.x - c1.y * c2.y, c1.x * c2.y + c2.x * c1.y };
}

__device__ float mag(Complex c) {
	return (float)sqrt((double)(c.x * c.x + c.y * c.y));
}
